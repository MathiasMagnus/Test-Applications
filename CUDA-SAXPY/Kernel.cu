#include "hip/hip_runtime.h"
#include <Kernel.hpp>


__global__
void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;

    if (i < n) y[i] = a * x[i] + y[i];
}

std::pair<cuda::event, cuda::event> saxpy(cuda::stream stream, std::size_t N, float a, cuda::array<float> x, cuda::array<float> y)
{
    return stream.record([&]()
    {
        saxpy<<<N, 256, 0, stream.get()>>>((int)N, a, x.data(), y.data());
    });
}
